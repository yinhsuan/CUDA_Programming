#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int maxIteration)
{
	float z_re = c_re, z_im = c_im;
	int i;
	for (i = 0; i < maxIteration; ++i)
	{
		if (z_re * z_re + z_im * z_im > 4.f)
		break;

		float new_re = z_re * z_re - z_im * z_im;
		float new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;
	}
	return i;
}


__global__ void mandelKernel(float lowerX, float lowerY, int resX, int resY, int maxIterations, float stepX, float stepY, int *device, int groupX, int groupY) {
    // To avoid error caused by the floating number, use the following pseudo code
    //

    // get the curreent thread location
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    int i, j;
    float x, y;
    for (i=thisX; i<thisX+groupX; i++) {
        for (j=thisY; j<thisY+groupY; j++) {
            x = lowerX + i * stepX;
            y = lowerY + j * stepY;
            device[thisY*resX + thisX] = mandel(x, y, maxIterations);
        }
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int threadsPerBlockX = 16;
    int threadsPerBlockY = 16;
    int groupX = 4;
    int groupY = 4;
    int size = resX * resY * sizeof(int);

    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // allocate memory space
    int* host;
    int* device;
    size_t pitch;
    hipHostAlloc(&host, size, hipHostMallocMapped);
    hipMallocPitch((void**)&device, &pitch, resX * sizeof(int), resY);

    // calculate
    dim3 block(threadsPerBlockX / groupX, threadsPerBlockY / groupY);
    dim3 grid(resX / block.x, resY / block.y);
    mandelKernel<<<grid, block>>>(lowerX, lowerY, resX, resY, maxIterations, stepX, stepY, *device, groupX, groupY);

    // copy
    hipMemcpy(host, device, size, hipMemcpyDeviceToHost); //  device(gpu) -> host(cpu)
    memcpy(img, host, size); // host -> img

    // release the memory
    free(host);
    hipFree(device);
}
